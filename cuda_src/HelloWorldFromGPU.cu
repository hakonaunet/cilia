// cudaParallelization.cu


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__ void helloFromGPU (void) {
    printf("Hello World from block %d, thread %d!\n", blockIdx.x, threadIdx.x);
}

int runHelloFromGPU(void) {
    helloFromGPU<<<2, 2>>>();
    hipDeviceSynchronize();

    // Check for errors in kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
    }

    return 0;
}

#ifdef RUN_HELLO_FROM_GPU
int main() {
    printf("Hello World from CPU!\n");
    runHelloFromGPU();
    return 0;
}
#endif